
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "itf/engine/filler.hpp"
#include "itf/engine/layer.hpp"
#include "itf/engine/util/im2col.hpp"
#include "itf/engine/util/math_functions.hpp"
#include "itf/engine/vision_layers.hpp"

namespace itf {

template <typename Dtype>
void CuDNNPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  CUDNN_CHECK(cudnnPoolingForward(handle_, pooling_desc_,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        top_desc_, top_data));
}

template <typename Dtype>
void CuDNNPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  CUDNN_CHECK(cudnnPoolingBackward(handle_, pooling_desc_,
        cudnn::dataType<Dtype>::one,
        top_desc_, top_data, top_desc_, top_diff,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        bottom_desc_, bottom_diff));
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNPoolingLayer);

}  // namespace itf
#endif
