#include "hip/hip_runtime.h"
#include "itf/trackers/buffgpu.h"
#include "itf/trackers/gpucommon.hpp"

#include <iostream>
template <typename ELEM_T>
MemBuff<ELEM_T>::MemBuff(int n, int c)
{
    elem_size=n;
    channel=c;
    byte_size=elem_size*channel*sizeof(ELEM_T);
    gpu_zalloc(d_data,byte_size,1);
    h_data =(ELEM_T *)zalloc(byte_size,1);
}
template <typename ELEM_T>
void MemBuff<ELEM_T>::SyncD2H()
{
    hipMemcpy(h_data,d_data,byte_size,hipMemcpyDeviceToHost);
}
template <typename ELEM_T>
void MemBuff<ELEM_T>::SyncD2HStream(hipStream_t& stream)
{
    hipMemcpyAsync(h_data,d_data,byte_size,hipMemcpyDeviceToHost,stream);
}
template <typename ELEM_T>
void MemBuff<ELEM_T>::SyncH2D()
{
    hipMemcpy(d_data,h_data,byte_size,hipMemcpyHostToDevice);
}
template <typename ELEM_T>
void MemBuff<ELEM_T>::SyncH2DStream(hipStream_t& stream)
{
    hipMemcpyAsync(d_data,h_data,byte_size,hipMemcpyHostToDevice,stream);
}
template <typename ELEM_T>
void MemBuff<ELEM_T>::updateGPU(ELEM_T* ptr)
{
    hipMemcpy(d_data,ptr,byte_size,hipMemcpyDeviceToDevice);
}
template <typename ELEM_T>
void MemBuff<ELEM_T>::updateCPU(ELEM_T* ptr)
{
    memcpy(h_data,ptr,byte_size);
}
template <typename ELEM_T>
void MemBuff<ELEM_T>::toZeroD()
{
    hipMemset(d_data,0,byte_size);
}
template <typename ELEM_T>
void MemBuff<ELEM_T>::toZeroH()
{
    memset(h_data,0,byte_size);
}
template <typename ELEM_T>
void MemBuff<ELEM_T>::copyFrom(MemBuff<ELEM_T>* src)
{
    hipMemcpy(d_data,src->gpu_ptr(),byte_size,hipMemcpyDeviceToDevice);
    memcpy(h_data,src->cpu_ptr(),byte_size);
}
template class MemBuff<float>;
template class MemBuff<int>;
template class MemBuff<FeatPts>;
template class MemBuff<TrkPts>;
template class MemBuff<float2>;
template class MemBuff<unsigned char>;
template class MemBuff<GroupTrack>;

__global__ void updateVecKernel(FeatPts* next_ptr,FeatPts* gpuBUff_Ptr,int* lenVec,int* status,int bufflen)
{
    int idx=threadIdx.x;
    int len = lenVec[idx];
    bool flag= status[idx];
    next_ptr[idx]=gpuBUff_Ptr[idx];
    lenVec[idx]=flag*(len+(len<bufflen));
}

void Tracks::init(int n,int l)
{
    nQue=n,buffLen=l,tailidx=0;
    trkData = new MemBuff<FeatPts>(nQue*buffLen);
    trkDataPtr=trkData->gpu_ptr();
    lenData = new MemBuff<int>(nQue);
    lenVec=lenData->gpu_ptr();
    veloData = new MemBuff<float2>(nQue*buffLen);
    veloDataPtr=veloData->gpu_ptr();
    distData = new MemBuff<float>(nQue*buffLen);
    distDataPtr=distData->gpu_ptr();
    curCpuPtr=trkData->cpu_ptr()+tailidx*nQue;
    spdData = new MemBuff<float>(nQue*buffLen);
    spdDataPtr=spdData->gpu_ptr();
    TracksInfo::init(n,l);
}

void Tracks::Sync()
{
    trkData->SyncD2H();
    lenData->SyncD2H();
}
void Group::init(int maxn,Tracks* trks)
{
    tracks=trks;
    trkPtsNum=tracks->nQue;
    trkPtsIdx = new MemBuff<int>(trkPtsNum*maxn);
    trkPtsIdxPtr=trkPtsIdx->gpu_ptr();
    ptsNum = new MemBuff<int>(maxn);
    ptsNumPtr=ptsNum->gpu_ptr();
    trkPts = new MemBuff<float2>(trkPtsNum*maxn);
    trkPtsPtr=trkPts->gpu_ptr();
    com = new MemBuff<float2>(maxn);
    comPtr=com->gpu_ptr();
    velo = new MemBuff<float2>(maxn);
    veloPtr=velo->gpu_ptr();
    bBox = new MemBuff<int>(maxn,4);
    bBoxPtr = bBox->gpu_ptr();
    polygon= new MemBuff<float2>(trkPtsNum*maxn);
    polygonPtr=polygon->gpu_ptr();
    polyCount = new MemBuff<int>(maxn);
    polyCountPtr=polyCount->gpu_ptr();
}
void Group::SyncD2H()
{
    trkPtsIdx->SyncD2H();
    ptsNum->SyncD2H();
    trkPts->SyncD2H();
    com->SyncD2H();
    velo->SyncD2H();
    bBox->SyncD2H();
    polygon->SyncD2H();
    polyCount->SyncD2H();
}
void Group::trkPtsSyncD2H()
{
    ptsNum->SyncD2H();
    trkPts->SyncD2H();
}
void Group::polySyncH2D()
{
    polygon->SyncH2D();
    polyCount->SyncH2D();
}
void Groups::init(int maxn,Tracks* trks)
{
    maxNumGroup=maxn;
    numGroups=0;
    Group::init(maxNumGroup,trks);
    /*
    tracks=trks;
    trkPtsNum=tracks->nQue;
    trkPtsIdx = new MemBuff<int>(trkPtsNum*maxNumGroup);
    trkPtsIdxPtr=trkPtsIdx->gpu_ptr();
    ptsNum = new MemBuff<int>(maxNumGroup);
    ptsNumPtr=ptsNum->gpu_ptr();
    trkPts = new MemBuff<float2>(trkPtsNum*maxNumGroup);
    trkPtsPtr=trkPts->gpu_ptr();
    com = new MemBuff<float2>(maxNumGroup);
    comPtr=com->gpu_ptr();
    velo = new MemBuff<float2>(maxNumGroup);
    veloPtr=velo->gpu_ptr();
    bBox = new MemBuff<int>(maxNumGroup,4);
    bBoxPtr = bBox->gpu_ptr();
    */
}
void GroupTrack::init(int maxn,Tracks* trks)
{
    buffLen=maxn;
    tailidx=0,len=0;
    Group::init(buffLen,trks);
}
void GroupTracks::addGroups(Groups* groups,int i)
{
    if(numGroup<maxNumGroup)
    {
        GroupTrack* nextGroup = getPtr(numGroup);
        nextGroup = new GroupTrack();
        nextGroup->init(buffLen,groups->tracks);
        GroupTrack* gpuPtr = groupTracks->gpu_ptr()+numGroup;
        hipMemcpy(gpuPtr,nextGroup,sizeof(GroupTrack),hipMemcpyHostToDevice);
        numGroup++;
    }
}
void GroupTracks::init(int maxn)
{
    numGroup=0,buffLen=10,maxNumGroup=maxn;
    groupTracks = new MemBuff<GroupTrack>(maxn);

}
